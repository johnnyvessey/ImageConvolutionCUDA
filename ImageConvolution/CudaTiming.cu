#include "CudaTiming.cuh"

CudaTiming::CudaTiming()
{
	check(hipEventCreate(&start));
	check(hipEventCreate(&stop));
	check(hipEventRecord(start, 0));
}

CudaTiming::~CudaTiming()
{
	check(hipEventDestroy(start));
	check(hipEventDestroy(stop));
}

void CudaTiming::Start()
{
	check(hipEventCreate(&start));
	check(hipEventCreate(&stop));
	check(hipEventRecord(start, 0));
}

void CudaTiming::Stop()
{
	check(hipEventRecord(stop, 0));
	check(hipEventSynchronize(stop));
}

void CudaTiming::PrintTime(const char* message)
{
	float elapsedTime;
	check(hipEventElapsedTime(&elapsedTime,
		start, stop));
	std::cout << message << ": " << elapsedTime << " ms\n";
}

float CudaTiming::GetTime()
{
	float elapsedTime;
	check(hipEventElapsedTime(&elapsedTime,
		start, stop));

	return elapsedTime;
}