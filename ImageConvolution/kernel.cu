#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <vector>

#include "lodepng.h"
#include "CudaTiming.cuh"

#include "hip/hip_runtime_api.h"
#include <hip/hip_fp16.h>
#include <random>

using std::vector;

#define CONV_SIDE_LENGTH 49

#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_Z 4


__constant__ float constantConv[CONV_SIDE_LENGTH * CONV_SIDE_LENGTH];

__device__ unsigned char clamp(float sum)
{
    //return (sum >= 255) * 255 + (sum > 0 && sum < 255) * static_cast<unsigned char>(sum);
    if (sum >= 255)
    {
        return 255;
    }
    else if (sum > 0)
    {
        return static_cast<unsigned char>(sum);
    }
    else {
        return 0;
    }
}
__global__ void Convolve(unsigned char* out, unsigned char* pixels, int width, int height)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int color = threadIdx.z;

    int col_offset = CONV_SIDE_LENGTH / 2;
    int row_offset = CONV_SIDE_LENGTH / 2;

    
        float sum = 0.0;

        for (int i = row - row_offset; i <= row + row_offset; i++)
        {
            if (i < 0 || i >= height)
                continue;
            for (int j = col - col_offset; j <= col + col_offset; j++)
            {
                if (j < 0 || j >= width)
                    continue;
                int convRow = i - (row - row_offset);
                int convCol = j - (col - col_offset);
                int convIdx = convRow * CONV_SIDE_LENGTH + convCol;


                int pixelIdx = (i * width + j) * 4 + color;

                unsigned char pixelVal = pixels[pixelIdx];
                sum = sum + (constantConv[convIdx] * pixelVal);
                

            }
        }
        int idx = (row * width + col) * 4 + color;
       

        out[idx] = clamp(sum);
    

}

__global__ void ConvolveSharedMemory(unsigned char* out, const unsigned char* pixels, const int width, const int height)
{

    int row = blockIdx.y * BLOCK_Y + threadIdx.y;
    int col = blockIdx.x * BLOCK_X + threadIdx.x;
    int color = threadIdx.z;

    int idx = (row * width + col) * 4 + color;
    if (idx >= width * height * BLOCK_Z) return;

    int convOffset = CONV_SIDE_LENGTH / 2;

    const int shared_block_width = BLOCK_X + CONV_SIDE_LENGTH - 1;
    const int shared_block_height = BLOCK_Y + CONV_SIDE_LENGTH - 1;
    const int shared_block_size = shared_block_width * shared_block_height;

    __shared__ unsigned char shared_block[shared_block_height * shared_block_width * BLOCK_Z];

    //set shared memory
    int sub_pixel_idx = threadIdx.y * blockDim.x + threadIdx.x;

    while (sub_pixel_idx < shared_block_size)
    {
        int x = sub_pixel_idx % shared_block_width;
        int y = sub_pixel_idx / shared_block_width;

        int x_global = (x - convOffset) + blockIdx.x * BLOCK_X;
        int y_global = (y - convOffset) + blockIdx.y * BLOCK_Y;

        if (x_global >= 0 && y_global >= 0 && x_global < width && y_global < height)
        {
            shared_block[(y * shared_block_width + x) * BLOCK_Z + color] = pixels[(y_global * width + x_global) * 4 + color];
        }
        else
        {
            shared_block[(y * shared_block_width + x) * BLOCK_Z + color] = 0;
        }

        sub_pixel_idx += BLOCK_X * BLOCK_Y;
    }

    __syncthreads();

    //set alpha to max value
    if (color == 3)
    {
        out[idx] = 255;
    }
    else
    {
        float sum = 0.0;

        for (int i = 0; i < CONV_SIDE_LENGTH; i++)
        {
            for (int j = 0; j < CONV_SIDE_LENGTH; j++)
            {
                int idx = ((threadIdx.y + i) * shared_block_width + threadIdx.x + j) * BLOCK_Z + color;
                sum += constantConv[i * CONV_SIDE_LENGTH + j] * shared_block[idx];
            }
        }
        
        out[idx] = clamp(sum);
    }


}


__global__ void ConvolveSharedMemory1D(unsigned char* out, unsigned char* pixels, int  width, int height)
{

    //int global_threadIdx = blockIdx.x * blockDim.x + threadIdx.x;

    //if (global_threadIdx >= width * height * BLOCK_Z) return;

    //int convOffset = CONV_SIDE_LENGTH / 2;

    //const int shared_block_width =  BLOCK_X + CONV_SIDE_LENGTH - 1;
    //const int shared_block_height = BLOCK_Y + CONV_SIDE_LENGTH - 1;
    //const int shared_block_size = shared_block_width * shared_block_height * BLOCK_Z;

    //int color = global_threadIdx % 4;
    //__shared__ unsigned char shared_block[shared_block_size];

    ////set shared memory
    //int sub_idx = threadIdx.x;

    //while (sub_idx < shared_block_size)
    //{
    //    int x = (sub_idx / 4) % shared_block_width;
    //    int y = (sub_idx / 4) / shared_block_width;

    //    int x_global = (x - convOffset) + blockIdx.x * BLOCK_X;
    //    int y_global = (y - convOffset) + blockIdx.y * BLOCK_Y;

    //    if (x_global >= 0 && y_global >= 0 && x_global < width && y_global < height)
    //    {
    //        shared_block[(x * shared_block_width + y) + color] = pixels[];
    //    }
    //    else
    //    {
    //        shared_block[sub_idx + color] = 0;
    //    }

    //    sub_idx += shared_block_size;
    //}

    //__syncthreads();

    ////set alpha to full value
    //if (color == 3)
    //{
    //    out[global_threadIdx] = 255;
    //}
    //else
    //{
    //    float sum = 0.0;

    //    int block_row = threadIdx.x / shared_block_width;
    //    int block_col = threadIdx.x / 
    //    int location_idx = threadIdx.x / 4;
    //    for (int i = 0; i < CONV_SIDE_LENGTH; i++)
    //    {
    //        for (int j = 0; j < CONV_SIDE_LENGTH; j++)
    //        {
    //            sum += constantConv[i * CONV_SIDE_LENGTH + j] * shared_block[];
    //        }
    //    }

    //    out[global_threadIdx] = clamp(sum);
    //}

}

vector<unsigned char> ConvolveImage(vector<unsigned char>& pixels, vector<float>& convolution, int width, int height, int convHeight, int convWidth)
{
    unsigned char* input;
    unsigned char* out;
    //float* cudaConv;


    int pixelCount = pixels.size();
    int pixelsMemory = sizeof(unsigned char) * pixelCount;

    check(hipMalloc((void**)&input, pixelsMemory));
    check(hipMalloc((void**)&out, pixelsMemory));
    //check(hipMalloc((void**)&cudaConv, convHeight * convWidth * sizeof(float)));

    //check(hipMemcpy(cudaConv, convolution.data(), convHeight * convWidth * sizeof(float), hipMemcpyHostToDevice));

    check(hipMemcpy(input, pixels.data(), pixelsMemory, hipMemcpyHostToDevice));

    dim3 pixelGrid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y);
    dim3 subGrid(BLOCK_X, BLOCK_Y, BLOCK_Z);

    const size_t num_iter = 10;
    float totalTime = 0.0f;
    for (size_t i = 0; i < num_iter; i++)
    {
        CudaTiming kernelTiming;
        kernelTiming.Start();
        ConvolveSharedMemory << < pixelGrid, subGrid >> > (out, input, width, height);
        //Convolve << < pixelGrid, subGrid >> > (out, input, width, height);
        hipProfilerStop();
        kernelTiming.Stop();
        totalTime += kernelTiming.GetTime();
    }

    std::cout << "Avg time: " << totalTime / (float)num_iter << " ms\n";


    unsigned char* outputPointer = (unsigned char*)malloc(pixelCount * sizeof(unsigned char));
    check(hipMemcpy(outputPointer, out, pixelCount * sizeof(unsigned char), hipMemcpyDeviceToHost));

    check(hipFree(input));
    //check(hipFree(cudaConv));

    vector<unsigned char> outputPixels;
    outputPixels.reserve(pixelCount);

    for (int i = 0; i < pixelCount; i++)
    {
        outputPixels.push_back(outputPointer[i]);
    }

    free(outputPointer);
    check(hipFree(out));

    return outputPixels;
}
int main(void) {
    vector<unsigned char> pixels;
    unsigned int width = 1024;
    unsigned int height = 1024;
    lodepng::decode(pixels, width, height, "cat_image.png");


    vector<float> convolution(CONV_SIDE_LENGTH * CONV_SIDE_LENGTH, 0);

    for (int i = 0; i < convolution.size(); i++)
    {
        convolution[i] = (2.0f / (float)(CONV_SIDE_LENGTH * CONV_SIDE_LENGTH)) * (float)rand() / (float)RAND_MAX;
    }
    //identity
    //convolution[(CONV_SIDE_LENGTH * CONV_SIDE_LENGTH) / 2 ] = 1.1;

    //convolution[25] = 2.3;
    //vector<float> convolution = { -1,-1,-1,-1,8,-1,-1,-1,-1 };

   
    check(hipMemcpyToSymbol(HIP_SYMBOL(constantConv), convolution.data(), convolution.size() * sizeof(float)));

    //CudaTiming ct;
   // ct.Start();
    vector<unsigned char> newImage = ConvolveImage(pixels, convolution, width, height, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH);
    //ct.Stop();
    //ct.PrintTime("Total function time");

    lodepng::encode("conv_image.png", newImage, width, height);

}