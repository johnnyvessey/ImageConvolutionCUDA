#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "lodepng.h"
#include <random>
#include "kernels.cuh"

#define PI 3.14159265


vector<float> generate_gaussian_blur(int sideLength, float sigma)
{
    vector<float> gaussian_blur_convolution(sideLength * sideLength, 0);

    float sum = 0.0f;
    for (size_t i = 0; i < sideLength; i++)
    {
        for (size_t j = 0; j < sideLength; j++)
        {
            int x = j - sideLength / 2;
            int y = i - sideLength / 2;
            float value = 1.0f / (2.0f * PI * sigma * sigma) * exp(-(x * x + y * y) / (2.0f * sigma * sigma));
            gaussian_blur_convolution[i * sideLength + j] = value;
            sum += value;
        }
    }

    for (size_t i = 0; i < sideLength * sideLength; i++)
    {
        gaussian_blur_convolution[i] /= sum;
    }
    return gaussian_blur_convolution;
}

//make sure convolution.size() is the same as CONV_SIDE_LENGTH * CONV_SIDE_LENGTH
void CreateConvolvedImage(vector<float> convolution, std::string inputFilename, std::string filename, bool optimized = true, bool naive = false, bool useConstantMemory = true, bool useChar = true)
{
    unsigned int width = IMAGE_WIDTH; //required to make variables because lodepng takes width + height as references
    unsigned int height = IMAGE_HEIGHT;
    vector<unsigned char> pixels;

    lodepng::decode(pixels, width, height, inputFilename);


    CudaTiming ct;
    ct.Start();
    
    unsigned char* newImage;
    if(optimized) newImage = ImageConvolution::ConvolveOptimized(pixels, convolution, IMAGE_WIDTH, IMAGE_HEIGHT, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH, useChar);
    else newImage = ImageConvolution::ConvolveImage(pixels, convolution, IMAGE_WIDTH, IMAGE_HEIGHT, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH, naive, useConstantMemory);

    ct.Stop();
    ct.PrintTime("Total function time");

    //this is just to make the pictures look better and easier to see the effects of the convolution visually (setting alpha to max value)
    for (size_t i = 3; i < 4 * IMAGE_WIDTH * IMAGE_HEIGHT; i += 4)
    {
        newImage[i] = 255;
    }

    lodepng::encode(filename, newImage, width, height);
    free(newImage);

}

void EdgeDetection(std::string inputFilename, std::string filename)
{
    
    unsigned int width = IMAGE_WIDTH; //required to make variables because lodepng takes width + height as references
    unsigned int height = IMAGE_HEIGHT;
    vector<unsigned char> pixels;

    lodepng::decode(pixels, width, height, inputFilename);

    //blur image slighly before applying sobel filters
    vector<float> blur = generate_gaussian_blur(CONV_SIDE_LENGTH, 2);
    unsigned char* blurImage = ImageConvolution::ConvolveOptimized(pixels, blur, IMAGE_WIDTH, IMAGE_HEIGHT, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH, false);
    for (size_t i = 0; i < pixels.size(); i++)
    {     
        pixels[i] = blurImage[i];
    }
    free(blurImage);

    vector<float> conv1 = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
    vector<float> conv2 = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };
    unsigned char* firstPass = ImageConvolution::ConvolveOptimized(pixels, conv1, IMAGE_WIDTH, IMAGE_HEIGHT, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH, false);

    for (size_t i = 0; i < pixels.size(); i++)
    {
        pixels[i] = firstPass[i];
    }
    free(firstPass);

    unsigned char* secondPass = ImageConvolution::ConvolveOptimized(pixels, conv2, IMAGE_WIDTH, IMAGE_HEIGHT, CONV_SIDE_LENGTH, CONV_SIDE_LENGTH, false);
    //this is just to make the pictures look better and easier to see the effects of the convolution visually (setting alpha to max value)
    for (size_t i = 3; i < 4 * IMAGE_WIDTH * IMAGE_HEIGHT; i += 4)
    {
        secondPass[i] = 255;
    }

    lodepng::encode(filename, secondPass, width, height);
    free(secondPass);


}


int main(void) {

    //large box blur
    vector<float> box_blur_convolution(CONV_SIDE_LENGTH * CONV_SIDE_LENGTH, 1.0f / (float)(CONV_SIDE_LENGTH * CONV_SIDE_LENGTH));

    vector<float> gaussian_blur_convolution = generate_gaussian_blur(CONV_SIDE_LENGTH, (float)CONV_SIDE_LENGTH / 4);

    std::cout << "Separable Optimized Convolution:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image_separable.png");


    //Add this to make the convolution non-separable
    gaussian_blur_convolution[0] = .2;

    std::cout << "\n------------------------\n\nNon-separable Optimized Convolution Int Shared Memory:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image_unseparable.png", true, false, true, false);

    std::cout << "\n------------------------\n\nNon-separable Optimized Convolution Char Shared Memory:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image_unseparable.png", true, false, true, true);


    std::cout << "\n------------------------\n\nBasic Shared Memory Convolution:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image.png", false, false);

    std::cout << "\n------------------------\n\nNaive Convolution with Constant Memory:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image.png", false, true);

    std::cout << "\n------------------------\n\nNaive Convolution with No Constant Memory:\n";
    CreateConvolvedImage(gaussian_blur_convolution, "cat_image_large.png", "conv_image.png", false, true, false);


    //EdgeDetection("cat_image.png", "sobel_image.png");

}